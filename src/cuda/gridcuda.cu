#include "hip/hip_runtime.h"
#include <gl/glew.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include <cutil.h>
//#include <cutil_gl_error.h>

#include <cudpp.h>

#include "../util.h"


////////////////////////////////////////////////////////////////////////////////
// Globals
////////////////////////////////////////////////////////////////////////////////
static CUDPPHandle g_accumPlan;
static CUDPPHandle g_maxPlan;

static GLuint g_inputBuffer = 0;
static GLuint g_outputBuffer = 0;

static float* gd_maxElement = NULL;

static int g_maxNumElements = 0;

//static int count = 0;


////////////////////////////////////////////////////////////////////////////////
// Entry point for Cuda functionality on host side
////////////////////////////////////////////////////////////////////////////////

bool equal( float* in, float* out, int size )
{
	for( int i = 0; i < size; ++i )
	{
		if( in[i] != out[i] )
			return false;
	}

	return true;
}

void dump( const char* name, float* ref, float* curr, int size )
{
	//fprintf( getLogFile(), "	dumping %s:\n", name );

	for( int i = 0; i < size; ++i )
	{
		//fprintf( getLogFile(), "	ref[%d]=%.1f, curr[%d]=%.1f\n", i, ref[i], i, curr[i] );
	}
}

extern "C" 
{
	void cudaDeleteScan()
	{
		cudppDestroyPlan( g_accumPlan );
		cudppDestroyPlan( g_maxPlan );

		if( gd_maxElement != NULL )
			hipFree( gd_maxElement );
	}

	void cudaInitScan( int maxNumElements )
	{
		// Accumulation
		CUDPPConfiguration config;
		config.algorithm = CUDPP_SCAN;
		config.op = CUDPP_ADD;
		config.datatype = CUDPP_FLOAT;
		config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;

		CUDPPResult result = cudppPlan( &g_accumPlan, config, maxNumElements, 1, 0 );

		if( result != CUDPP_SUCCESS )
		{
	        //fprintf( getLogFile(), "Error creating CUDPPPlan, aborting execution...\n" );
			exit(1);
		}

 		// Reduction
		config.algorithm = CUDPP_SCAN;
		config.op = CUDPP_MAX;
		config.datatype = CUDPP_FLOAT;
		config.options = CUDPP_OPTION_BACKWARD | CUDPP_OPTION_INCLUSIVE;

		result = cudppPlan( &g_maxPlan, config, maxNumElements, 1, 0 );

		if( result != CUDPP_SUCCESS )
		{
	        //fprintf( getLogFile(), "Error creating CUDPPPlan, aborting execution...\n" );
			exit(1);
		}

 		unsigned int byteTotal = maxNumElements * sizeof(float);
 		CUDA_SAFE_CALL( hipMalloc( (void**)&gd_maxElement, byteTotal ) );
 		CUT_CHECK_ERROR( "hipMalloc failed.\n" );

		g_maxNumElements = maxNumElements;



//count = 0;
	}

	void cudaRegisterInputBuffer( GLuint bufferId )
	{
		//if( g_inputBuffer > 0 )
			//CUDA_SAFE_CALL( cudaGLUnregisterBufferObject( g_inputBuffer ) );

		// we already rgistered with gridsort.cu
		//CUDA_SAFE_CALL( cudaGLRegisterBufferObject( bufferId ) );
		//CUT_CHECK_ERROR( "cudaGLRegisterBufferObject failed.\n" );
		g_inputBuffer = bufferId;
	}

	void cudaRegisterOutputBuffer( GLuint bufferId )
	{
		//if( g_outputBuffer > 0 )
			//CUDA_SAFE_CALL( cudaGLUnregisterBufferObject( g_outputBuffer ) );

		// we already rgistered with gridsort.cu
		//CUDA_SAFE_CALL( cudaGLRegisterBufferObject( bufferId ) );
		//CUT_CHECK_ERROR( "cudaGLRegisterBufferObject failed.\n" );
		g_outputBuffer = bufferId;
	}

	void cudaUnregisterBuffers()
	{
		if( g_inputBuffer > 0 )
			CUDA_SAFE_CALL( cudaGLUnregisterBufferObject( g_inputBuffer ) );
		if( g_outputBuffer > 0 )
			CUDA_SAFE_CALL( cudaGLUnregisterBufferObject( g_outputBuffer ) );
	}

	bool cudaRunScan( int numElements, float* maxCount, float* maxAccum )
	{
		//if( numElements > g_maxNumElements )
		//{
			//fprintf( getLogFile(), "  cuda numElements %d greater than maxNumElements %d\n", numElements, g_maxNumElements );
			//return false;
		//}

		float* d_inputBuffer;
		float* d_outputBuffer;
		
		// Map in/out buffers
		//fprintf( getLogFile(), "  cuda map pbo's %d e %d\n", g_inputBuffer, g_outputBuffer );

		CUDA_SAFE_CALL( cudaGLMapBufferObject( (void**)&d_inputBuffer, g_inputBuffer ) );
		CUDA_SAFE_CALL( cudaGLMapBufferObject( (void**)&d_outputBuffer, g_outputBuffer ) );

		CUT_CHECK_ERROR( "cudaGLMapBufferObject failed.\n" );

		//fprintf( getLogFile(), "  cuda run accum\n" );



//static float* inputRefEven = new float[g_maxNumElements];
//static float* outputRefEven = new float[g_maxNumElements];
//static float* inputRefOdd = new float[g_maxNumElements];
//static float* outputRefOdd = new float[g_maxNumElements];
//static float* mappedInEven = NULL;
//static float* mappedOutEven = NULL;
//static float* mappedInOdd = NULL;
//static float* mappedOutOdd = NULL;
//
//float* tmpin = new float[g_maxNumElements];
//float* tmpout = new float[g_maxNumElements];
//
//
//if( count == 0 )
//{
//	mappedInEven = d_inputBuffer;
//	mappedOutEven = d_outputBuffer;
//
//	std::fill( inputRefEven, inputRefEven + g_maxNumElements, 0.0f );
//
//	CUDA_SAFE_CALL( hipMemcpy( &inputRefEven[0], &d_inputBuffer[0], sizeof(float) * numElements - 1, hipMemcpyDeviceToHost ) );
//}
//else if( count == 1 )
//{
//	mappedInOdd = d_inputBuffer;
//	mappedOutOdd = d_outputBuffer;
//
//	std::fill( inputRefOdd, inputRefOdd + g_maxNumElements, 0.0f );
//
//	CUDA_SAFE_CALL( hipMemcpy( &inputRefOdd[0], &d_inputBuffer[0], sizeof(float) * numElements - 1, hipMemcpyDeviceToHost ) );
//}
//else if( count % 2 == 0 )
//{
//	if( mappedInEven != d_inputBuffer )
//	{
//		//fprintf( getLogFile(), "  cuda pointers changed since last call!\n" );
//		return false;
//	}
//
//	if( mappedOutEven != d_outputBuffer )
//	{
//		//fprintf( getLogFile(), "  cuda pointers changed since last call!\n" );
//		return false;
//	}
//
//	std::fill( tmpin, tmpin + g_maxNumElements, 0.0f );
//	CUDA_SAFE_CALL( hipMemcpy( &tmpin[0], &d_inputBuffer[0], sizeof(float) * numElements - 1, hipMemcpyDeviceToHost ) );
//	if( !equal( inputRefEven, tmpin, numElements - 1 ) )
//	{
//		//fprintf( getLogFile(), "  cuda buffers changed since last call!\n" );
//		dump( "input", inputRefEven, tmpin, numElements - 1 );
//		dump( "output", outputRefEven, tmpout, numElements );
//		return false;
//	}
//
//}
//else
//{
//	if( mappedInOdd != d_inputBuffer )
//	{
//		//fprintf( getLogFile(), "  cuda pointers changed since last call!\n" );
//		return false;
//	}
//
//	if( mappedOutOdd != d_outputBuffer )
//	{
//		//fprintf( getLogFile(), "  cuda pointers changed since last call!\n" );
//		return false;
//	}
//
//	std::fill( tmpin, tmpin + g_maxNumElements, 0.0f );
//	CUDA_SAFE_CALL( hipMemcpy( &tmpin[0], &d_inputBuffer[0], sizeof(float) * numElements - 1, hipMemcpyDeviceToHost ) );
//	if( !equal( inputRefOdd, tmpin, numElements - 1 ) )
//	{
//		//fprintf( getLogFile(), "  cuda buffers changed since last call!\n" );
//		dump( "input", inputRefOdd, tmpin, numElements - 1 );
//		dump( "output", outputRefOdd, tmpout, numElements );
//		return false;
//	}
//
//}







		
		// Accumulate
		CUDPPResult result = cudppScan( g_accumPlan, d_outputBuffer, d_inputBuffer, numElements );

		//if( result != CUDPP_SUCCESS )
		//{
			//fprintf( getLogFile(), "  - error in cuda accum: %d\n", result );
		//}

		CUT_CHECK_ERROR( "Scan execution failed.\n" );







//if( count == 0 )
//{
//	std::fill( outputRefEven, outputRefEven + g_maxNumElements, 0.0f );
//
//	CUDA_SAFE_CALL( hipMemcpy( &outputRefEven[0], &d_outputBuffer[0], sizeof(float) * numElements, hipMemcpyDeviceToHost ) );
//}
//else if( count == 1 )
//{
//	std::fill( outputRefOdd, outputRefOdd + g_maxNumElements, 0.0f );
//
//	CUDA_SAFE_CALL( hipMemcpy( &outputRefOdd[0], &d_outputBuffer[0], sizeof(float) * numElements, hipMemcpyDeviceToHost ) );
//}
//else if( count % 2 == 0 )
//{
//	std::fill( tmpout, tmpout + g_maxNumElements, 0.0f );
//	CUDA_SAFE_CALL( hipMemcpy( &tmpout[0], &d_outputBuffer[0], sizeof(float) * numElements, hipMemcpyDeviceToHost ) );
//	if( !equal( outputRefEven, tmpout, numElements ) )
//	{
//		//fprintf( getLogFile(), "  cuda buffers changed since last call!\n" );
//		dump( "input", inputRefEven, tmpin, numElements - 1 );
//		dump( "output", outputRefEven, tmpout, numElements );
//		return false;
//	}
//}
//else
//{
//	std::fill( tmpout, tmpout + g_maxNumElements, 0.0f );
//	CUDA_SAFE_CALL( hipMemcpy( &tmpout[0], &d_outputBuffer[0], sizeof(float) * numElements, hipMemcpyDeviceToHost ) );
//	if( !equal( outputRefOdd, tmpout, numElements ) )
//	{
//		//fprintf( getLogFile(), "  cuda buffers changed since last call!\n" );
//		dump( "input", inputRefOdd, tmpin, numElements - 1 );
//		dump( "output", outputRefOdd, tmpout, numElements );
//		return false;
//	}
//}
//
//
//
//
//delete tmpin;
//delete tmpout;
//
//++count;










		// not needed when using single-pass fragment programs
		//if( maxCount != NULL )
		//{
			////fprintf( getLogFile(), "  cuda run max count\n" );

			//// Find maximum count
			//result = cudppScan( g_maxPlan, gd_maxElement, d_inputBuffer, numElements - 1 );

			//if( result != CUDPP_SUCCESS )
			//{
			//	//fprintf( getLogFile(), "  - error in cuda max count: %d\n", result );	
			//}

			//CUT_CHECK_ERROR( "Scan execution failed.\n" );

			//// Retrieve maximum count
			//CUDA_SAFE_CALL( hipMemcpy( maxCount, &gd_maxElement[0], sizeof(float), hipMemcpyDeviceToHost ) );
			//CUT_CHECK_ERROR( "hipMemcpy failed.\n" );
		//}

		if( maxAccum != NULL )
		{
			//fprintf( getLogFile(), "  cuda get max accum\n" );

			// Retrieve maximum
			CUDA_SAFE_CALL( hipMemcpy( maxAccum, &d_outputBuffer[numElements-1], sizeof(float), hipMemcpyDeviceToHost ) );
			CUT_CHECK_ERROR( "hipMemcpy failed.\n" );
		}

		//fprintf( getLogFile(), "  cuda unmap pbo's\n" );

		// Unmap in/out buffers
		CUDA_SAFE_CALL( cudaGLUnmapBufferObject( g_outputBuffer ) );
		CUDA_SAFE_CALL( cudaGLUnmapBufferObject( g_inputBuffer ) );

		CUT_CHECK_ERROR( "cudaGLUnmapBufferObject failed.\n" );

		return true;
	}

} // extern "C"
