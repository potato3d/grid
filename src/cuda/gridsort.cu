#include <gl/glew.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include <cutil.h>
//#include <cutil_gl_error.h>

#include "radixsort.cuh"

#include "../util.h"

////////////////////////////////////////////////////////////////////////////////
// Globals
////////////////////////////////////////////////////////////////////////////////
static GLuint g_gridBuffer0 = 0;
static GLuint g_gridBuffer1 = 0;

////////////////////////////////////////////////////////////////////////////////
// Entry point for Cuda functionality on host side
////////////////////////////////////////////////////////////////////////////////

extern "C" 
{
	void cudaRegisterGridBuffer0( GLuint bufferId )
	{
		CUDA_SAFE_CALL( cudaGLRegisterBufferObject( bufferId ) );
		CUT_CHECK_ERROR( "cudaGLRegisterBufferObject failed.\n" );
		g_gridBuffer0 = bufferId;
	}

	void cudaRegisterGridBuffer1( GLuint bufferId )
	{
		CUDA_SAFE_CALL( cudaGLRegisterBufferObject( bufferId ) );
		CUT_CHECK_ERROR( "cudaGLRegisterBufferObject failed.\n" );
		g_gridBuffer1 = bufferId;
	}

	void dump( const char* name, float* curr, int size )
	{
		//fprintf( getLogFile(), "	dumping %s:\n", name );

		for( int i = 0; i < size; ++i )
		{
			//fprintf( getLogFile(), "	v[%d]=%.1f\n", i, curr[i] );
		}
	}

	void cudaRunSort( uint numElements )
	{
		float* d_grid0;
		float* d_grid1;
		
		// Map in/out buffers
		//fprintf( getLogFile(), "  cuda map pbo's %d e %d\n", g_gridBuffer0, g_gridBuffer1 );

		CUDA_SAFE_CALL( cudaGLMapBufferObject( (void**)&d_grid0, g_gridBuffer0 ) );
		CUDA_SAFE_CALL( cudaGLMapBufferObject( (void**)&d_grid1, g_gridBuffer1 ) );
		
		//unsigned int* grid0 = (unsigned int*)d_grid0;
		//unsigned int* grid1 = (unsigned int*)d_grid1;
		
		// Sort
		//fprintf( getLogFile(), "  cuda run sort\n" );

		//dump( "sort input", d_grid0, numElements * 2 );
		//exit( 1 );

	    RadixSort( (KeyValuePair*) d_grid0, (KeyValuePair*) d_grid1, numElements, 32);
		CUT_CHECK_ERROR( "Sort execution failed.\n" );
		
		// TODO: testing
		//int numBytes = numElements * 2 * sizeof(float);
		//float* testout = (float*)malloc( numBytes );
		//CUDA_SAFE_CALL( hipMemcpy( testout, d_grid0, numBytes, hipMemcpyDeviceToHost ) );
		//CUT_CHECK_ERROR( "hipMemcpy failed.\n" );
		
		// Unmap grid buffers
		//fprintf( getLogFile(), "  cuda unmap pbo's\n" );

		CUDA_SAFE_CALL( cudaGLUnmapBufferObject( g_gridBuffer1 ) );
 		CUDA_SAFE_CALL( cudaGLUnmapBufferObject( g_gridBuffer0 ) );
	}

} // extern "C"
